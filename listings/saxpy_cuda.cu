
#include <hip/hip_runtime.h>
#define ARRAY_SIZE 1024 
#define BLOCK_SIZE 256

static float x[ARRAY_SIZE];
static float y[ARRAY_SIZE];
static float a = 3.0;

// the identifier __global__ indicates that it should be launched on the GPU
__global__ 
void saxpy(int N, float d_a, float* d_x, float* d_y) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i < N) d_y[i] = d_a*d_x[i] + d_y[i];
}	

int main() {

	// initialize memory on CPU
	for (int i = 0; i < ARRAY_SIZE; ++i) {
        	x[i] = 1.0; y[i] = 2.0;
        }

	// allocate memory on GPU
	float *d_x, *d_y;
	hipMalloc(&d_x, ARRAY_SIZE*sizeof(float));
	hipMalloc(&d_y, ARRAY_SIZE*sizeof(float));

	// copy data to GPU
	hipMemcpy(d_x, x, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	
	// launch kernel
	int nblocks = (ARRAY_SIZE + BLOCK_SIZE) / BLOCK_SIZE;
	saxpy<<<nblocks,BLOCK_SIZE>>>(ARRAY_SIZE, a, d_x, d_y);

	// copy back
	hipMemcpy(y, d_y, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);

	// free up memory on GPU
	hipFree(d_x);
	hipFree(d_y);

}


